#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools_GPU.h"
#include "DomaineMath_GPU.h"

#include "FractalMath.h"
#include "JuliaMath.h"
#include "MandelbrotMath.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__device__ void fractal(uchar4* ptrDevPixels, uint width, uint height, DomaineMath domaineMath, uint n, FractalMath* fractalMath);

__global__ void julia(uchar4* ptrDevPixels, uint width, uint height, DomaineMath domaineMath, uint n);
__global__ void mandelbrot(uchar4* ptrDevPixels, uint width, uint height, DomaineMath domaineMath, uint n);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__device__ void fractal(uchar4* ptrDevPixels, int w, int h, DomaineMath mathDomain, int n, FractalMath* fractalMath)
    {
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    uchar4 color;

    // math
    double x;
    double y;
    // screen
    int pixelI;
    int pixelJ;

    int s = TID;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)

	mathDomain.toXY(pixelI, pixelJ, &x, &y); //  (i,j) -> (x,y)

	fractalMath->colorXY(&color, x, y, n); // update color

	ptrDevPixels[s] = color;

	s += NB_THREAD;
	}
    }

__global__ void julia(uchar4* ptrDevPixels, int width, int height, DomaineMath domaineMath, int n)
    {
    float c1 = -0.12;
    float c2 = 0.85;
    FractalMath* fractalMath = new JuliaMath(c1, c2);

    fractal(ptrDevPixels, width, height, domaineMath, n, fractalMath);

    delete fractalMath;
    }

__global__ void mandelbrot(uchar4* ptrDevPixels, int width, int height, DomaineMath domaineMath, int n)
    {
    FractalMath* fractalMath = new MandelbrotMath();

    fractal(ptrDevPixels, width, height, domaineMath, n, fractalMath);

    delete fractalMath;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

