#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Device.h"
#include "Fractal.h"
using namespace gpu;



/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void julia(uchar4* ptrDevPixels, int w, int h, DomaineMath domaineMath, int n);
extern __global__ void mandelbrot(uchar4* ptrTabPixels, int w, int h, DomaineMath domaineMath, int n);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/
Fractal::Fractal(const Grid& grid, uint w, uint h, int nMin, int nMax, float dt, const DomaineMath& domaineMath, bool isJulia):
	Animable_I<uchar4>(grid, w, h, "Fractal_Cuda_Julia_Mandelbrot", domaineMath),
	variateurAnimation(Interval<float>(nMin, nMax), 1)
    {
    this->n = nMax;
    this->dt = dt;
    this->whoProcess = isJulia ? &Fractal::processFractalJulia : &Fractal::processFractalMandelbrot;
    }

Fractal::~Fractal()
    {
   // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void Fractal::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    (this->*whoProcess)(ptrDevPixels, w, h, domaineMath);
    }


void Fractal::processFractalJulia(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    julia<<<this->dg, this->db>>>(ptrDevPixels, w, h, domaineMath, this->n);
    }

void Fractal::processFractalMandelbrot(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    mandelbrot<<<this->dg, this->db>>>(ptrDevPixels, w, h, domaineMath, this->n);
    }

/**
 * Override
 * Call periodicly by the API
 */
void Fractal::animationStep()
    {
    this->n = variateurAnimation.varierAndGet(); // in [0,2pi]
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

