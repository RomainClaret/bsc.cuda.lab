#include "hip/hip_runtime.h"
#include "RayTracing.h"

#include <iostream>
#include <assert.h>

#include "Device.h"
#include "AleaTools.h"

using namespace gpu;



/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/
RayTracing::RayTracing(uint w, uint h)
    {
    this->w = w;
    this->h = h;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/
void RayTracing::populateSpheres(int nbSpheres, Sphere *ptrSpheres)
    {
    // Populate spheres
    AleaTools aleaTools = AleaTools();
    float bord = 200.0f;

    float rayon;
    float3 centre;
    float hue;
    for (int i = 0; i < nbSpheres; i++)
	{
	rayon = aleaTools.uniformeAB(20, w/10);

	centre.x = aleaTools.uniformeAB(bord, h-bord);
	centre.y = aleaTools.uniformeAB(bord, w-bord);
	centre.z = aleaTools.uniformeAB(10, 2*w);

	hue = aleaTools.uniforme01();

	ptrSpheres[i] = Sphere(centre, rayon, hue);
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

