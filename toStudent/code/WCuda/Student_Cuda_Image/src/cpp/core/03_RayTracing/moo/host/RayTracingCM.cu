#include "hip/hip_runtime.h"
#include "RayTracingCM.h"

#include <iostream>
#include <assert.h>

#include "Device.h"
#include "AleaTools.h"
#include "ConstantMemoryLink.h"

using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/
#define NB_SPHERE 50
//__constant__ Sphere CM_DATA[NB_SPHERE];

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/
extern __global__ void rayTracingCM(uchar4* ptrDevPixels, int w, int h, int nbSpheres, int t, Sphere* ptrSpheres);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
ConstantMemoryLink constantMemoryLink(void)
{
    Sphere* ptrDevCML;
    size_t currentSize = NB_SPHERE * sizeof(Sphere);
//    HANDLE_ERROR(hipGetSymbolAddress((void **) &ptrDevCM, CM_DATA));
    ConstantMemoryLink cmLink;
    cmLink.n = NB_SPHERE;
    cmLink.ptrDevTab = (void**) ptrDevCML;
    cmLink.sizeAll = currentSize;
    return cmLink;
}

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/
RayTracingCM::RayTracingCM(const Grid& grid, uint w, uint h, float dt, int spheres):
	Animable_I<uchar4>(grid, w, h, "RayTracing_CM_Cuda")
    {
    this->dt = dt;
    this->nbSpheres = nbSpheres;
    this->ptrSpheres = new Sphere[this->nbSpheres];

    this->t = 0.0;

    this->ptrSpheres = new Sphere[NB_SPHERE];

    AleaTools aleaTools = AleaTools();
    float bord = 200.0f;

    float rayon;
    float3 centre;
    float hue;

    for(int i=0;i<NB_SPHERE;i++)
	{
	rayon = aleaTools.uniformeAB(20, w/10);

	centre.x = aleaTools.uniformeAB(bord, h-bord);
	centre.y = aleaTools.uniformeAB(bord, w-bord);
	centre.z = aleaTools.uniformeAB(10, 2*w);

	hue = aleaTools.uniforme01();

	ptrSpheres[i] = *(new Sphere(centre, rayon, hue));
	}

    ConstantMemoryLink cmLink = constantMemoryLink();
    ptrDevCML = (Sphere*)cmLink.ptrDevTab;
    size_t currentSize = cmLink.sizeAll;
    HANDLE_ERROR(hipMemcpy(ptrDevCML, ptrSpheres, currentSize, hipMemcpyHostToDevice));

    }

RayTracingCM::~RayTracingCM()
    {
//    delete[] CM_DATA;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void RayTracingCM::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    rayTracingCM<<<dg,db>>>(ptrDevPixels, w, h, NB_SPHERE, this->t, ptrDevCML);
    }

/**
 * Override
 * Call periodicly by the API
 */
void RayTracingCM::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

