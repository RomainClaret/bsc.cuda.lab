#include "hip/hip_runtime.h"
#include <iostream>
#include "Indice1D.h"
#include "Indice2D.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/
__global__ void histogramDevice(int* ptrDevInput, int* ptrDevOut, int n, int size);

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
__global__ void histogramDevice(int* ptrDevInput, int* ptrDevOut, int n, int size)
    {
    extern __shared__ int tabData[];

    const int THREADS = Indice1D::tid();

    int s = Indice2D::nbThreadX();

    if(Indice1D::tidLocal() < size)
	{
	tabData[Indice1D::tidLocal()] = 0;
	}

    __syncthreads();

    while(s < n)
	{
	atomicAdd(&tabData[ptrDevInput[s]], 1);
	s += THREADS;
	}

    __syncthreads();

    if(Indice1D::tidLocal() < size)
	{
	atomicAdd(&ptrDevOut[Indice1D::tidLocal()], tabData[Indice1D::tidLocal()]);
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

