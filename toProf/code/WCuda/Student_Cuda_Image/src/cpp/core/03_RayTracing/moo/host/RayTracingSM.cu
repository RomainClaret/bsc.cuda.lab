#include "hip/hip_runtime.h"
#include "RayTracingSM.h"

#include <iostream>
#include <assert.h>

#include "Device.h"
#include "AleaTools.h"

using namespace gpu;



/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/
extern __global__ void rayTracingSM(uchar4* ptrDevPixels, int w, int h, int spheres, int t, Sphere* ptrSpheres);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/
RayTracingSM::RayTracingSM(const Grid& grid, uint w, uint h, float dt, int spheres):
	Animable_I<uchar4>(grid, w, h, "RayTracing_SM_Cuda"),
	RayTracing(w,h)
    {
    this->dt = dt;
    this->nbSpheres = nbSpheres;
    this->ptrSpheres = new Sphere[this->nbSpheres];

    this->t = 0.0;

    RayTracing::populateSpheres(this->nbSpheres, this->ptrSpheres);
    }

RayTracingSM::~RayTracingSM()
    {
    delete[] ptrSpheres;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void RayTracingSM::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Sphere* ptrDevSphere=NULL;
    size_t currentSize = this->nbSpheres * sizeof(Sphere);

    HANDLE_ERROR(hipMalloc(&ptrDevSphere,currentSize));
    HANDLE_ERROR(hipMemcpy(ptrDevSphere, this->ptrSpheres, currentSize, hipMemcpyHostToDevice));

    rayTracingSM<<<dg,db>>>(ptrDevPixels, w, h, this->nbSpheres, this->t, ptrDevSphere);

    HANDLE_ERROR(hipFree(ptrDevSphere));
    }

/**
 * Override
 * Call periodicly by the API
 */
void RayTracingSM::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

