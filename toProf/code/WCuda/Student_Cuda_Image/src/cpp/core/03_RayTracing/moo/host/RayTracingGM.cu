#include "hip/hip_runtime.h"
#include "RayTracingGM.h"

#include <iostream>
#include <assert.h>

#include "Device.h"
#include "AleaTools.h"

using namespace gpu;



/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/
extern __global__ void rayTracingGM(uchar4* ptrDevPixels, int w, int h, int spheres, int t, Sphere* ptrSpheres);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/
RayTracingGM::RayTracingGM(const Grid& grid, uint w, uint h, float dt, int nbSpheres):
	Animable_I<uchar4>(grid, w, h, "RayTracing_GM_Cuda"),
	RayTracing(w,h)
    {
    this->dt = dt;
    this->nbSpheres = nbSpheres;
    this->ptrSpheres = new Sphere[this->nbSpheres];

    this->t = 0.0;

    RayTracing::populateSpheres(this->nbSpheres, this->ptrSpheres);
    }

RayTracingGM::~RayTracingGM()
    {
    delete[] ptrSpheres;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void RayTracingGM::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Sphere* ptrDevSphere=NULL;
    size_t currentSize = this->nbSpheres * sizeof(Sphere);

    HANDLE_ERROR(hipMalloc(&ptrDevSphere, currentSize));
    HANDLE_ERROR(hipMemcpy(ptrDevSphere, this->ptrSpheres, currentSize, hipMemcpyHostToDevice));

    rayTracingGM<<<dg,db>>>(ptrDevPixels, w, h, this->nbSpheres, this->t, ptrDevSphere);

    HANDLE_ERROR(hipFree(ptrDevSphere));
    }

/**
 * Override
 * Call periodicly by the API
 */
void RayTracingGM::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

