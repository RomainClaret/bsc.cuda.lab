#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools_GPU.h"
#include "ConstantMemoryLink.h"

#include "Sphere.h"
#include "RayTracingMath.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
__global__ void rayTracingGM(uchar4* ptrDevPixels, int w, int h, int spheres, int t, Sphere* ptrSpheres);
__global__ void rayTracingCM(uchar4* ptrDevPixels, int w, int h, int spheres, int t, Sphere* ptrSpheres);
__global__ void rayTracingSM(uchar4* ptrDevPixels, int w, int h, int spheres, int t, Sphere* ptrSpheres);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
static __device__ void work(uchar4* ptrDevPixels, int w, int h, int spheres, int t, Sphere* ptrSpheres);
static __device__ void gm2sm(Sphere* ptrSpheresGM, Sphere* ptrSpheresSM, int size);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
    __global__
    void rayTracingGM(uchar4* ptrDevPixels, int w, int h, int spheres, int t, Sphere* ptrSpheres)
	{
	work(ptrDevPixels, w, h, spheres, t, ptrSpheres);
	}

    __global__
    void rayTracingCM(uchar4* ptrDevPixels, int w, int h, int spheres, int t, Sphere* ptrSpheres)
	{
	work(ptrDevPixels, w, h, spheres, t, ptrSpheres);
	}

    __global__
    void rayTracingSM(uchar4* ptrDevPixels, int w, int h, int spheres, int t, Sphere* ptrSpheres)
	{
	extern __shared__ Sphere ptrSpheresSM[];

	gm2sm(ptrSpheres, ptrSpheresSM, spheres);

	__syncthreads();
	work(ptrDevPixels, w, h, spheres, t, ptrSpheresSM);
	}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

    __device__
    void work(uchar4* ptrDevPixels, int w, int h, int spheres, int t, Sphere* ptrSpheres)
	{
	RayTracingMath rayTracingMath = RayTracingMath(t);

	const int TID = Indice2D::tid();
	const int NB_THREAD = Indice2D::nbThread();
	const int WH = w*h;

	int i;
	int j;

	int s = TID;
	while (s < WH)
	    {
	    IndiceTools::toIJ(s, w, &i, &j);

	    rayTracingMath.colorIJ(&ptrDevPixels[s], i, j, t, ptrSpheres, spheres);

	    s += NB_THREAD;
	    }
	}

    __device__
    void gm2sm(Sphere* ptrSpheresGM, Sphere* ptrSpheresSM, int size)
	{
	const int TID_LOCAL = Indice2D::tidLocal();
	const int NB_THREAD_BLOCK = Indice2D::nbThreadBlock();

	int s = TID_LOCAL;
	while (s < size)
	    {
	    ptrSpheresSM[s] = ptrSpheresGM[s];
	    s += NB_THREAD_BLOCK;
	    }
	}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

